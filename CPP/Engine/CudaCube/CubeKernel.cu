#include "hip/hip_runtime.h"
/*******************************************************************************
 FILE         :   CubeKernel.cu 

 COPYRIGHT    :   DMAlex, 2011

 DESCRIPTION  :   Kernel (GPU) for Cube 

 PROGRAMMED BY:   Alex Fedosov

 CREATION DATE:   07/22/2011

 LAST UPDATE  :   07/22/2011
*******************************************************************************/



#include <hip/hip_runtime.h>         

#include "Cube.h"
#include "BasicCube.h"

 
using namespace Its;

__device__  SizeIt GetIndexX()
{
  return blockIdx.x * blockDim.x + threadIdx.x; 
}

__device__  SizeIt GetIndexY()
{
  return blockIdx.y * blockDim.y + threadIdx.y; 
}


extern "C" __global__ void CubeEntry( int gf, int random,
                                      void* dataIn,  void* dataInU, void* dataInD, 
                                      void* dataOut, int sizeX, int sizeY, Probe* probe ) 
{
  PointIt size      ( sizeX       , sizeY       );
  PointIt index     ( GetIndexX() , GetIndexY() );
  PointIt position  ( threadIdx.x , threadIdx.y );

  BasicCube cube( dataIn, dataInU, dataInD,                    
                  size,   index,   position
                  );


  __syncthreads();

  cube.It( dataOut, gf, random, probe );
}

